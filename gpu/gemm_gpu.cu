#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_C_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// =======================================================
// o1：简单并行化，每个线程负责计算 C 中的一个元素
// =======================================================
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float* C, int M, int N, int K) {
    // 计算当前线程负责的输出矩阵 C 中的行和列
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        // 累加计算 C[row, col] 的值
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K) {
    // 设置线程块尺寸为 16x16，保证足够的并行度
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                  (M + blockSize.y - 1) / blockSize.y);
    gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// =======================================================
// o2：基于 tiling 的优化，利用共享内存提升数据复用
// =======================================================
__global__ void gemm_gpu_o2_kernel(float* A, float* B, float* C, int M, int N, int K) {
    // 定义 tile 尺寸，可根据硬件特性调优
    const int TILE_SIZE = 16;
    // 分配共享内存
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    // 计算全局索引
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;

    // 遍历所有 tile
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // 将 A 的 tile 加载到共享内存（边界检查）
        if (row < M && (t * TILE_SIZE + threadIdx.x) < K)
            tile_A[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;
        
        // 将 B 的 tile 加载到共享内存（边界检查）
        if (col < N && (t * TILE_SIZE + threadIdx.y) < K)
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;
        
        __syncthreads();
        
        // 对当前 tile 内进行乘累加
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // 将计算结果写回全局内存
    if (row < M && col < N)
        C[row * N + col] = sum;
}

void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K) {
    const int TILE_SIZE = 16;
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE,
                  (M + TILE_SIZE - 1) / TILE_SIZE);
    gemm_gpu_o2_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// =======================================================
// o3：进一步调优版本，在 tiling 基础上尝试较大 tile 尺寸并利用 unroll 提升性能
// =======================================================
__global__ void gemm_gpu_o3_kernel(float* A, float* B, float* C, int M, int N, int K) {
    // 这里选用更大的 tile 尺寸，视 GPU 资源而定
    const int TILE_SIZE = 32;
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // 加载 A tile
        if (row < M && (t * TILE_SIZE + threadIdx.x) < K)
            tile_A[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;
        
        // 加载 B tile
        if (col < N && (t * TILE_SIZE + threadIdx.y) < K)
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;
        
        __syncthreads();
        
        // 使用 unroll 优化内层循环
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N)
        C[row * N + col] = sum;
}

void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K) {
    const int TILE_SIZE = 32;
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE,
                  (M + TILE_SIZE - 1) / TILE_SIZE);
    gemm_gpu_o3_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}
